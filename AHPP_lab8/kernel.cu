#include "hip/hip_runtime.h"
//#include <iostream>
//#include <opencv2/core/core.hpp>
//#include <opencv2/highgui/highgui.hpp>
//#include <stdio.h>
//#include <stdlib.h>
//
//#define FIBER 32
//
//using namespace std;
//using namespace cv;
//
//void cudaProcess(const Mat &image, Point &size);
//void cudaInitMem(const Mat &image, uchar *outputImg, Point size);
//
//__global__ void kernel(Point* size, unsigned* data) {
//
//    int bx = blockIdx.x;
//    int by = blockIdx.y;
//    int tx = threadIdx.x;
//    int ty = threadIdx.y;
//    int dx = blockDim.x;
//    int dy = blockDim.y;
//
//    int block_x = bx * dx;
//    int block_y = by * dy;
//
//    int i = block_x + tx;
//    int j = block_y + ty;
//
//    int X = size->x;
//    int Y = size->y;
//
//    int idx = j * X + i;
//    int idx_reverse = (Y - 1 - j) * X + (X - 1 - i);
//
//    __shared__ unsigned smem[(FIBER*3)*FIBER];
//
//    const unsigned inputWidth = size->x / sizeof(int);
//
//    const int baseX = (blockIdx.x*blockDim.x+threadIdx.x)*3;
//    const int baseY = blockIdx.y * blockDim.y + threadIdx.y;
//
////    smem[threadIdx.y * FIBER * 3 + threadIdx.x] = data[baseY * inputWidth + baseX];
////    smem[threadIdx.y * FIBER * 3 + threadIdx.x + blockDim.x] = data[baseY * inputWidth + baseX + blockDim.x];
////    smem[threadIdx.y * FIBER * 3 + threadIdx.x + blockDim.x*2] = data[baseY * inputWidth+baseX + blockDim.x * 2];
////
////    __syncthreads();
//
////    unsigned buffer[3];
//    idx *= 3;
//    idx_reverse += 3;
//
//    if (j < Y / 2) {
////        buffer[0] = data[idx];
////        buffer[1] = data[idx + 1];
////        buffer[2] = data[idx + 2];
//
//        data[idx] = 0;
//        data[idx + 1] = 180;
//        data[idx + 2] = 0;
//
////        data[idx_reverse] = buffer[0];
////        data[idx_reverse + 1] = buffer[1];
////        data[idx_reverse + 2] = buffer[2];
//    }
//
//
////    else if ((j == Y / 2) && (i <= X / 2) && (Y % 2 != 0)) {
////        idx_reverse = j * X + (X - 1 - i);
////        buffer = data[idx];
////        data[idx] = data[idx_reverse];
////        data[idx_reverse] = buffer;
////    }
//
//
////    uchar buffer = 0;
////
////    if (j < Y / 2) {
////        buffer = data[idx];
////        data[idx] = data[idx_reverse];
////        data[idx_reverse] = buffer;
////    }
////    else if ((j == Y / 2) && (i <= X / 2) && (Y % 2 != 0)) {
////        idx_reverse = j * X + (X - 1 - i);
////        buffer = data[idx];
////        data[idx] = data[idx_reverse];
////        data[idx_reverse] = buffer;
////    }
//}
//
//int main(int argc, char** argv) {
//
//    if(argc != 2) {
//        cout <<" Usage: display_image ImageToLoadAndDisplay" << endl;
//        return -1;
//    }
//
//    Mat image;
//    image = imread(argv[1], CV_LOAD_IMAGE_COLOR);
//
//    if(! image.data ) {
//        cerr <<  "Could not open or find the image" << std::endl ;
//        return -1;
//    }
//
//    Point size = Point_<int>(image.cols, image.rows);
//
//    uchar* output = (uchar*)malloc(image.cols * image.rows * sizeof(char));
//    cudaInitMem(image, output, size);
//
////    cudaProcess(image, size);
//
//
//
//    waitKey(0);
//    return 0;
//}
//
////void cudaProcess(const Mat &image, Point &size) {
////    Point* cuda_size;
////    uchar* cuda_data;
////    size_t data_size = sizeof(uchar) * size.x * size.y;
////
////    hipError_t error = hipMalloc((void**) &cuda_data, data_size);
////    if (error != hipSuccess) {
////        cerr << hipGetErrorString(error) << endl;
////    }
////
////    error = hipMemcpy(cuda_data, image.data, data_size, hipMemcpyHostToDevice);
////    if (error != hipSuccess) {
////        cerr << hipGetErrorString(error) << endl;
////    }
////
////    error = hipMalloc((void**) &cuda_size, sizeof(Point));
////    if (error != hipSuccess) {
////        cerr << hipGetErrorString(error) << endl;
////    }
////
////    error = hipMemcpy(cuda_size, &size, sizeof(Point), hipMemcpyHostToDevice);
////    if (error != hipSuccess) {
////        cerr << hipGetErrorString(error) << endl;
////    }
////
////    dim3 threads(FIBER, FIBER);
////    dim3 blocks((size.x + (FIBER - 1)) / FIBER, (size.y + ( FIBER - 1)) / FIBER);
////    kernel <<< blocks, threads >>> (cuda_size, cuda_data);
////
////    error = hipMemcpy(image.data, cuda_data, data_size, hipMemcpyDeviceToHost);
////    if (error != hipSuccess) {
////        cerr << hipGetErrorString(error) << endl;
////    }
////}
//
//void cudaInitMem(const Mat &image, uchar* outputImg, Point size) {
//    Point* cuda_size;
//    unsigned *cuda_data = NULL;
//
//    int pitch = 0;
//    int width = size.x * 3;
//    hipError_t error = hipMallocPitch((void **)&cuda_data, (size_t *) & pitch, width, size.y);
//    if (error != hipSuccess) {
//        cerr << hipGetErrorString(error) << endl;
//    }
//    error = hipMemcpy2D(cuda_data, pitch, image.data, width, width, size.y, hipMemcpyHostToDevice);
//    if (error != hipSuccess) {
//        cerr << hipGetErrorString(error) << endl;
//    }
//    // можем работать с cuda_data, при этом ширина – pitch байт
//
//    // hipMemcpy2D(outputImg, width, cuda_data, pitch, width, height, hipMemcpyDeviceToHost);
//    // в outputImg ширина изображения снова width байт
//
//    error = hipMalloc((void**) &cuda_size, sizeof(Point));
//    if (error != hipSuccess) {
//        cerr << hipGetErrorString(error) << endl;
//    }
//
//    error = hipMemcpy(cuda_size, &size, sizeof(Point), hipMemcpyHostToDevice);
//    if (error != hipSuccess) {
//        cerr << hipGetErrorString(error) << endl;
//    }
//
//    dim3 threads(FIBER, FIBER);
//    dim3 blocks((width + (FIBER - 1)) / FIBER, (size.y + ( FIBER - 1)) / FIBER);
//    kernel <<< blocks, threads >>> (cuda_size, cuda_data);
//
//
//    error = hipMemcpy2D(image.data, width, cuda_data, pitch, width, size.y, hipMemcpyDeviceToHost);
//    if (error != hipSuccess) {
//        cerr << hipGetErrorString(error) << endl;
//    }
//
//    namedWindow( "Display window", WINDOW_AUTOSIZE );
//    imshow( "Display window", image );
//}

#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <stdio.h>
#include <stdlib.h>

#define FIBER 32

using namespace std;
using namespace cv;

void cudaProcess(const Mat &image, Point &size);

__global__ void kernel(Point* size, uchar* data) {
    __shared__ unsigned smem[(FIBER*3) * FIBER];
    uchar buffer[3];

    int pitch = ( ( size->x * 3 + 127 )  / 128 ) * 128;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int dx = blockDim.x;
    int dy = blockDim.y;

    int block_x = bx * dx;
    int block_y = by * dy;

    int i = block_x + tx;
    int j = block_y + ty;

    int X = size->x;
    int Y = size->y;

    int idx = (j * pitch + i * 3);
    int idx_reverse = (Y - 1 - j) * pitch + (X - 1 - i) * 3;

    const int baseX = i * 3;

    smem[ty * dx * 3 + tx] = data[j * pitch / 4 + baseX];
    smem[ty * dx * 3 + tx + dx] = data[j * pitch / 4 + baseX + dx];
    smem[ty * dx * 3 + tx + dx * 2] = data[j * pitch / 4 + baseX + dx * 2];

    __syncthreads();

    if (j < Y / 2 && i <= size->x) {
        buffer[0] = data[idx];
        buffer[1] = data[idx + 1];
        buffer[2] = data[idx + 2];

        data[idx] = data[idx_reverse];
        data[idx + 1] = data[idx_reverse + 1];
        data[idx + 2] = data[idx_reverse + 2];

        data[idx_reverse] = buffer[0];
        data[idx_reverse + 1] = buffer[1];
        data[idx_reverse + 2] = buffer[2];
    }
    if ((j == Y / 2) && (i <= X / 2) && (Y % 2 != 0)) {
        idx_reverse = j * X + (X - 3 - i);
        buffer[0] = data[idx];
        buffer[1] = data[idx + 1];
        buffer[2] = data[idx + 2];

        data[idx] = data[idx_reverse];
        data[idx + 1] = data[idx_reverse + 1];
        data[idx + 2] = data[idx_reverse + 2];

        data[idx_reverse] = buffer[0];
        data[idx_reverse + 1] = buffer[1];
        data[idx_reverse + 2] = buffer[2];
    }
}

int main(int argc, char** argv) {

    if(argc != 2) {
        cout <<" Usage: display_image ImageToLoadAndDisplay" << endl;
        return -1;
    }

    Mat image, imageOld;
    imageOld = imread(argv[1], CV_LOAD_IMAGE_COLOR);
    image = imread(argv[1], CV_LOAD_IMAGE_COLOR);

    if(! image.data ) {
        cerr <<  "Could not open or find the image" << std::endl ;
        return -1;
    }

    Point size = Point_<int>(image.cols, image.rows);

    cudaProcess(image, size);

    namedWindow( "New image", WINDOW_AUTOSIZE );
    imshow( "New image", image );

    namedWindow( "Old image", WINDOW_AUTOSIZE );
    imshow( "Old image", imageOld );

    waitKey(0);
    return 0;
}

void cudaProcess(const Mat &image, Point &size) {
    Point* cuda_size;
    uchar* cuda_data;

    int width = size.x * 3;
    int height = size.y;

    const int pitch = ( ( width + 127 )  / 128 ) * 128;
    hipMalloc( (void **)&cuda_data, pitch * height );

    for (int i = 0; i < height; i++) {
        hipMemcpy(
                &cuda_data[i * pitch],
                &image.data[i * width],
                width,
                hipMemcpyHostToDevice);
    }

    hipError_t error = hipMalloc((void**) &cuda_size, sizeof(Point));
    if (error != hipSuccess) {
        cerr << hipGetErrorString(error) << endl;
    }

    // t1
    error = hipMemcpy(cuda_size, &size, sizeof(Point), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        cerr << hipGetErrorString(error) << endl;
    }

    // t2
    dim3 threads(FIBER, FIBER);
    dim3 blocks((pitch + (FIBER - 1)) / FIBER, (size.y + ( FIBER - 1)) / FIBER);
    kernel <<< blocks, threads >>> (cuda_size, cuda_data);
    
    //t3
    error = hipMemcpy2D(image.data, width, cuda_data, pitch, width, height, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        cerr << hipGetErrorString(error) << endl;
    }
    // t4
}