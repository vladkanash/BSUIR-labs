#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <stdio.h>
#include <stdlib.h>

#define FIBER 32

using namespace std;
using namespace cv;

void cudaProcess(const Mat &image, Point &size);

__global__ void kernel(Point* size, uchar* data) {

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int dx = blockDim.x;
    int dy = blockDim.y;

    int block_x = bx * dx;
    int block_y = by * dy;

    int i = block_x + tx;
    int j = block_y + ty;

    int X = size->x;
    int Y = size->y;

    int idx = j * X + i;
    int idx_reverse = (Y - 1 - j) * X + (X - 1 - i);

    uchar buffer = 0;

    if (j < Y / 2) {
        buffer = data[idx];
        data[idx] = data[idx_reverse];
        data[idx_reverse] = buffer;
    }
    else if ((j == Y / 2) && (i <= X / 2) && (Y % 2 != 0)) {
        idx_reverse = j * X + (X - 1 - i);
        buffer = data[idx];
        data[idx] = data[idx_reverse];
        data[idx_reverse] = buffer;
    }
}

int main(int argc, char** argv) {

    if(argc != 2) {
        cout <<" Usage: display_image ImageToLoadAndDisplay" << endl;
        return -1;
    }

    Mat image;
    image = imread(argv[1], CV_LOAD_IMAGE_GRAYSCALE);

    if(! image.data ) {
        cerr <<  "Could not open or find the image" << std::endl ;
        return -1;
    }

    Point size = Point_<int>(image.cols, image.rows);

    cudaProcess(image, size);

    namedWindow( "Display window", WINDOW_AUTOSIZE );
    imshow( "Display window", image );

    waitKey(0);
    return 0;
}

void cudaProcess(const Mat &image, Point &size) {
    Point* cuda_size;
    uchar* cuda_data;
    size_t data_size = sizeof(uchar) * size.x * size.y;

    hipError_t error = hipMalloc((void**) &cuda_data, data_size);
    if (error != hipSuccess) {
        cerr << hipGetErrorString(error) << endl;
    }

    error = hipMemcpy(cuda_data, image.data, data_size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        cerr << hipGetErrorString(error) << endl;
    }

    error = hipMalloc((void**) &cuda_size, sizeof(Point));
    if (error != hipSuccess) {
        cerr << hipGetErrorString(error) << endl;
    }

    error = hipMemcpy(cuda_size, &size, sizeof(Point), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        cerr << hipGetErrorString(error) << endl;
    }

    dim3 threads(FIBER, FIBER);
    dim3 blocks((size.x + (FIBER - 1)) / FIBER, (size.y + ( FIBER - 1)) / FIBER);
    kernel <<< blocks, threads >>> (cuda_size, cuda_data);

    error = hipMemcpy(image.data, cuda_data, data_size, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        cerr << hipGetErrorString(error) << endl;
    }
}