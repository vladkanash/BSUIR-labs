#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>


#define FIBER 32
#define N 1024
#define DATA_SIZE N * N * sizeof(int)

void print(int** r);

__global__ void kernel_shared(int *a, int *c, int *b, int *r) {
    __shared__ int smem[FIBER * FIBER];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int dx = blockDim.x;
    int dy = blockDim.y;

    int block_x = bx * dx;
    int block_y = by * dy;

    int x = block_x + tx;
    int y = block_y + ty;
    int idx = x + y * N;

    int smem_x = by * dy + tx;
    int smem_y = bx * dx + ty;
    int smem_idx = smem_x + smem_y * N;

    smem[dx * ty + tx] = a[smem_idx] + a[smem_idx];

    __syncthreads();

	r[idx] = smem[dy * tx + ty] * b[idx] - c[idx];
}

__global__ void kernel(int *a, int *x, int *b, int *r) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int dx = blockDim.x;
    int dy = blockDim.y;

    int i = bx * dx + tx;
    int j = by * dy + ty;
    r[i+ j * N] = (a[j + i * N] + a[j + i * N]) * b[i + j * N] - x[i + j * N];
}

using namespace std;

int* processMtrx(int* A, int* B, int* C) {
    int *R = (int*)aligned_alloc(32, DATA_SIZE);
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            R[i * N + j] = (A[j * N + i] + A[j * N + i]) * B[i * N + j] - C[i * N + j];
        }
    }
    return R;
}

void myCudaMalloc(int **ptr) {
    hipError_t error = hipMalloc((void**) ptr, DATA_SIZE);
    if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}
}

void cudaMemcpyHost2Device(int *src, int *dst) {
    hipError_t error = hipMemcpy(dst, src, DATA_SIZE, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}
}

void cudaMemcpyDevice2Host(int *src, int *dst) {
    hipError_t error = hipMemcpy(src, dst, DATA_SIZE, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}
}

bool checkForErrors(int *ptr1, int *ptr2, int *ptr3) {
    for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			if((ptr1[i * N + j] != ptr3[i * N + j]) || (ptr1[i * N + j] != ptr2[i * N + j])) {
                printf("\n%d != %d [%d]\n", ptr1[i * N + j], ptr2[i * N + j], i * N + j);
                return false;
            }

    return true;
}

int* randMtrx(int *MATR)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            MATR[i * N + j] = rand() % 3;
        }
    }

    return MATR;
}

void print(int* r)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%d\t", r[i * N + j]);
            if (j == N - 1)
            {
                printf("\n");
            }
        }
    }
    printf("\n");
    printf("\n");
}

int* processCPU(int *A, int *B, int *X) {
    int * R ;

    randMtrx(A);
    randMtrx(X);
    randMtrx(B);

//    printf("original =============\n");
//    print(A);
//    print(B);
//    printf("==================================\n");

    struct timeval stopm, startm;
    gettimeofday(&startm, NULL);

    R = processMtrx(A, B, X);

    gettimeofday(&stopm, NULL);

    printf("runTimeCPU =  %f \n", (float)(stopm.tv_usec - startm.tv_usec) / 1000);
    return R;
}

int* processGPU(int *A, int *B, int *X, bool shared) {
    int* R;
    int *Res = (int*)aligned_alloc(32, DATA_SIZE);
    memset(Res, 0, DATA_SIZE);

    myCudaMalloc(&R);
    cudaMemcpyHost2Device(Res, R);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    dim3 threads(FIBER, FIBER);
    dim3 blocks((N + (FIBER - 1)) / FIBER, (N + ( FIBER - 1)) / FIBER);

    hipEventSynchronize(start);

    if (shared) {
        kernel_shared <<< blocks, threads >>> (A, X, B, R);
    } else {
        kernel <<< blocks, threads >>> (A, X, B, R);
    }

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("%s\n", hipGetErrorString(error));
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float timer = 0;

    hipEventElapsedTime(&timer, start, stop);
    if (shared) {
        cout << "runTimeGPU shared = " << timer << endl;
    } else {
        cout << "runTimeGPU = " << timer << endl;
    }
    hipEventRecord(start);

    cudaMemcpyDevice2Host(Res, R);
    return Res;
}

int main(int argc, char* argv[])
{
    int *dev_A_shared, *dev_B_shared, *dev_C_shared;
    int *dev_A, *dev_B, *dev_C;
    int *A, *B, *C;
    int *cpu_result, *gpu_result, *gpu_result_shared;

	A = (int*)aligned_alloc(32, DATA_SIZE);
	B = (int*)aligned_alloc(32, DATA_SIZE);
	C = (int*)aligned_alloc(32, DATA_SIZE);

    cpu_result = processCPU(A, B, C);

    myCudaMalloc(&dev_A_shared);
    myCudaMalloc(&dev_B_shared);
    myCudaMalloc(&dev_C_shared);

    myCudaMalloc(&dev_A);
    myCudaMalloc(&dev_B);
    myCudaMalloc(&dev_C);


    cudaMemcpyHost2Device(A, dev_A_shared);
    cudaMemcpyHost2Device(B, dev_B_shared);
    cudaMemcpyHost2Device(C, dev_C_shared);

    cudaMemcpyHost2Device(A, dev_A);
    cudaMemcpyHost2Device(B, dev_B);
    cudaMemcpyHost2Device(C, dev_C);


    gpu_result = processGPU(dev_A, dev_B, dev_C, false);
    gpu_result_shared = processGPU(dev_A_shared, dev_B_shared, dev_C_shared, true);

    if (!checkForErrors(cpu_result, gpu_result_shared, gpu_result)) {
        printf("Errors occured!\n");
    } else {
        printf("No errors!\n");
    }


//    print(gpu_result_shared);
//    printf("\n\n");
//    print(cpu_result);

}

